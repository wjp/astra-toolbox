#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include <cstdio>
#include <cassert>
#include <iostream>


typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_FanProjTexture;


namespace astraCUDA {

const unsigned int g_anglesPerBlock = 16;
const unsigned int g_blockSliceSize = 32;
const unsigned int g_blockSlices = 16;

const unsigned int g_MaxAngles = 2560;

__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];


static bool bindProjDataTexture(float* data, unsigned int pitch, unsigned int width, unsigned int height, hipTextureAddressMode mode = hipAddressModeBorder)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_FanProjTexture.addressMode[0] = mode;
	gT_FanProjTexture.addressMode[1] = mode;
	gT_FanProjTexture.filterMode = hipFilterModeLinear;
	gT_FanProjTexture.normalized = false;

	hipBindTexture2D(0, gT_FanProjTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);

	// TODO: error value?

	return true;
}

__global__ void devFanBP(float* D_volData, unsigned int volPitch, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = - ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	// TODO: Distance correction?

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float fSrcX = gC_SrcX[angle];
		const float fSrcY = gC_SrcY[angle];
		const float fDetSX = gC_DetSX[angle];
		const float fDetSY = gC_DetSY[angle];
		const float fDetUX = gC_DetUX[angle];
		const float fDetUY = gC_DetUY[angle];

		const float fXD = fSrcX - fX;
		const float fYD = fSrcY - fY;

		const float fNum = fDetSY * fXD - fDetSX * fYD + fX*fSrcY - fY*fSrcX;
		const float fDen = fDetUX * fYD - fDetUY * fXD;
		
		const float fT = fNum / fDen;
		fVal += tex2D(gT_FanProjTexture, fT, fA);
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

// supersampling version
__global__ void devFanBP_SS(float* D_volData, unsigned int volPitch, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fXb = ( X - 0.5f*dims.iVolWidth + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);
	const float fYb = - ( Y - 0.5f*dims.iVolHeight + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);

	const float fSubStep = 1.0f/dims.iRaysPerPixelDim;

	float* volData = (float*)D_volData;

	fOutputScale /= (dims.iRaysPerPixelDim * dims.iRaysPerPixelDim);

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	// TODO: Distance correction?

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float fSrcX = gC_SrcX[angle];
		const float fSrcY = gC_SrcY[angle];
		const float fDetSX = gC_DetSX[angle];
		const float fDetSY = gC_DetSY[angle];
		const float fDetUX = gC_DetUX[angle];
		const float fDetUY = gC_DetUY[angle];

		// TODO: Optimize these loops...
		float fX = fXb;
		for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
			float fY = fYb;
			for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
				const float fXD = fSrcX - fX;
				const float fYD = fSrcY - fY;

				const float fNum = fDetSY * fXD - fDetSX * fYD + fX*fSrcY - fY*fSrcX;
				const float fDen = fDetUX * fYD - fDetUY * fXD;
		
				const float fT = fNum / fDen;
				fVal += tex2D(gT_FanProjTexture, fT, fA);
				fY -= fSubStep;
			}
			fX += fSubStep;
		}
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}


// BP specifically for SART.
// It includes (free) weighting with voxel weight.
// It assumes the proj texture is set up _without_ padding, unlike regular BP.
__global__ void devFanBP_SART(float* D_volData, unsigned int volPitch, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = - ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	// TODO: Distance correction?

	// TODO: Constant memory vs parameters.
	const float fSrcX = gC_SrcX[0];
	const float fSrcY = gC_SrcY[0];
	const float fDetSX = gC_DetSX[0];
	const float fDetSY = gC_DetSY[0];
	const float fDetUX = gC_DetUX[0];
	const float fDetUY = gC_DetUY[0];

	const float fXD = fSrcX - fX;
	const float fYD = fSrcY - fY;

	const float fNum = fDetSY * fXD - fDetSX * fYD + fX*fSrcY - fY*fSrcX;
	const float fDen = fDetUX * fYD - fDetUY * fXD;
		
	const float fT = fNum / fDen;
	const float fVal = tex2D(gT_FanProjTexture, fT, 0.5f);

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

// Weighted BP for use in fan beam FBP
// Each pixel/ray is weighted by 1/L^2 where L is the distance to the source.
__global__ void devFanBP_FBPWeighted(float* D_volData, unsigned int volPitch, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = - ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	// TODO: Distance correction?

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float fSrcX = gC_SrcX[angle];
		const float fSrcY = gC_SrcY[angle];
		const float fDetSX = gC_DetSX[angle];
		const float fDetSY = gC_DetSY[angle];
		const float fDetUX = gC_DetUX[angle];
		const float fDetUY = gC_DetUY[angle];

		const float fXD = fSrcX - fX;
		const float fYD = fSrcY - fY;

		const float fNum = fDetSY * fXD - fDetSX * fYD + fX*fSrcY - fY*fSrcX;
		const float fDen = fDetUX * fYD - fDetUY * fXD;

		const float fWeight = fXD*fXD + fYD*fYD;
		
		const float fT = fNum / fDen;
		fVal += tex2D(gT_FanProjTexture, fT, fA) / fWeight;
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}


bool FanBP_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {
		if (dims.iRaysPerPixelDim > 1)
			devFanBP_SS<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, dims, fOutputScale);
		else
			devFanBP<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, dims, fOutputScale);
	}
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipStreamDestroy(stream);

	return true;
}

bool FanBP_FBPWeighted_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {
		devFanBP_FBPWeighted<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, dims, fOutputScale);
	}
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipStreamDestroy(stream);

	return true;
}

// D_projData is a pointer to one padded sinogram line
bool FanBP_SART(float* D_volumeData, unsigned int volumePitch,
                float* D_projData, unsigned int projPitch,
                unsigned int angle,
                const SDimensions& dims, const SFanProjection* angles,
                float fOutputScale)
{
	// only one angle
	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, 1, hipAddressModeClamp);

	// transfer angle to constant memory
#define TRANSFER_TO_CONSTANT(name) do { hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), &(angles[angle].f##name), sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);

#undef TRANSFER_TO_CONSTANT

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	devFanBP_SART<<<dimGrid, dimBlock>>>(D_volumeData, volumePitch, dims, fOutputScale);
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	return true;
}

bool FanBP(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanBP_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle, fOutputScale);
		if (!ret)
			return false;
	}
	return true;
}

bool FanBP_FBPWeighted(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanBP_FBPWeighted_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle, fOutputScale);

		if (!ret)
			return false;
	}
	return true;
}


}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_projData;

	SDimensions dims;
	dims.iVolWidth = 128;
	dims.iVolHeight = 128;
	dims.iProjAngles = 180;
	dims.iProjDets = 256;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;
	unsigned int volumePitch, projPitch;

	SFanProjection projs[180];

	projs[0].fSrcX = 0.0f;
	projs[0].fSrcY = 1536.0f;
	projs[0].fDetSX = 128.0f;
	projs[0].fDetSY = -512.0f;
	projs[0].fDetUX = -1.0f;
	projs[0].fDetUY = 0.0f;

#define ROTATE0(name,i,alpha) do { projs[i].f##name##X = projs[0].f##name##X * cos(alpha) - projs[0].f##name##Y * sin(alpha); projs[i].f##name##Y = projs[0].f##name##X * sin(alpha) + projs[0].f##name##Y * cos(alpha); } while(0)

	for (int i = 1; i < 180; ++i) {
		ROTATE0(Src, i, i*2*M_PI/180);
		ROTATE0(DetS, i, i*2*M_PI/180);
		ROTATE0(DetU, i, i*2*M_PI/180);
	}

#undef ROTATE0

	allocateVolume(D_volumeData, dims.iVolWidth, dims.iVolHeight, volumePitch);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_projData, dims.iProjDets, dims.iProjAngles, projPitch);
	printf("pitch: %u\n", projPitch);

	unsigned int y, x;
	float* sino = loadImage("sino.png", y, x);

	float* img = new float[dims.iVolWidth*dims.iVolHeight];

	memset(img, 0, dims.iVolWidth*dims.iVolHeight*sizeof(float));

	copyVolumeToDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);
	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_projData, projPitch);

	FanBP(D_volumeData, volumePitch, D_projData, projPitch, dims, projs, 1.0f);

	copyVolumeFromDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);

	saveImage("vol.png",dims.iVolHeight,dims.iVolWidth,img);

	return 0;
}
#endif
