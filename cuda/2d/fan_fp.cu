#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>


typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_FanVolumeTexture;


namespace astraCUDA {

static const unsigned g_MaxAngles = 2560;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];


// optimization parameters
static const unsigned int g_anglesPerBlock = 16;
static const unsigned int g_detBlockSize = 32;
static const unsigned int g_blockSlices = 64;

static bool bindVolumeDataTexture(float* data, hipArray*& dataArray, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	dataArray = 0;
	hipMallocArray(&dataArray, &channelDesc, width, height);
	hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	gT_FanVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_FanVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_FanVolumeTexture.filterMode = hipFilterModeLinear;
	gT_FanVolumeTexture.normalized = false;

	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.
	//hipBindTexture2D(0, gT_FanVolumeTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);
	hipBindTextureToArray(gT_FanVolumeTexture, dataArray, channelDesc);

	// TODO: error value?

	return true;
}

// projection for angles that are roughly horizontal
// (detector roughly vertical)
__global__ void FanFPhorizontal(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	float* projData = (float*)D_projData;
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	const int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;
	if (angle >= endAngle)
		return;

	const int detector = blockIdx.y * g_detBlockSize + relDet;

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fDetSX = gC_DetSX[angle];
	const float fDetSY = gC_DetSY[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];

	float fVal = 0.0f;

	const float fdx = fabsf(fDetSX + detector*fDetUX + 0.5f - fSrcX);
	const float fdy = fabsf(fDetSY + detector*fDetUY + 0.5f - fSrcY);

	if (fdy > fdx)
		return;


	for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
		const float fDet = detector + (0.5f + iSubT) / dims.iRaysPerDet;

		const float fDetX = fDetSX + fDet * fDetUX;
		const float fDetY = fDetSY + fDet * fDetUY;

		// ray: y = alpha * x + beta
		const float alpha = (fSrcY - fDetY) / (fSrcX - fDetX);
		const float beta = fSrcY - alpha * fSrcX;
	
		const float fDistCorr = sqrt(alpha*alpha+1.0f) * outputScale / dims.iRaysPerDet;

		// intersect ray with first slice

		float fY = -alpha * (startSlice - 0.5f*dims.iVolWidth + 0.5f) - beta + 0.5f*dims.iVolHeight - 0.5f + 0.5f;
		float fX = startSlice + 0.5f;

		int endSlice = startSlice + g_blockSlices;
		if (endSlice > dims.iVolWidth)
			endSlice = dims.iVolWidth;

		float fV = 0.0f;
		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fV += tex2D(gT_FanVolumeTexture, fX, fY);
			fY -= alpha;
			fX += 1.0f;
		}

		fVal += fV * fDistCorr;

	}

	projData[angle*projPitch+detector] += fVal;
}


// projection for angles that are roughly vertical
// (detector roughly horizontal)
__global__ void FanFPvertical(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	const int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;

	if (angle >= endAngle)
		return;

	const int detector = blockIdx.y * g_detBlockSize + relDet;

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	float* projData = (float*)D_projData;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fDetSX = gC_DetSX[angle];
	const float fDetSY = gC_DetSY[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];

	float fVal = 0.0f;

	const float fdx = fabsf(fDetSX + detector*fDetUX + 0.5f - fSrcX);
	const float fdy = fabsf(fDetSY + detector*fDetUY + 0.5f - fSrcY);

	if (fdy <= fdx)
		return;


	for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
		const float fDet = detector + (0.5f + iSubT) / dims.iRaysPerDet /*- gC_angle_offset[angle]*/;

		const float fDetX = fDetSX + fDet * fDetUX;
		const float fDetY = fDetSY + fDet * fDetUY;

		// ray: x = alpha * y + beta
		const float alpha = (fSrcX - fDetX) / (fSrcY - fDetY);
		const float beta = fSrcX - alpha * fSrcY;
	
		const float fDistCorr = sqrt(alpha*alpha+1) * outputScale / dims.iRaysPerDet;

		// intersect ray with first slice

		float fX = -alpha * (startSlice - 0.5f*dims.iVolHeight + 0.5f) + beta + 0.5f*dims.iVolWidth - 0.5f + 0.5f;
		float fY = startSlice + 0.5f;

		int endSlice = startSlice + g_blockSlices;
		if (endSlice > dims.iVolHeight)
			endSlice = dims.iVolHeight;

		float fV = 0.0f;

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fV += tex2D(gT_FanVolumeTexture, fX, fY);
			fX -= alpha;
			fY += 1.0f;
		}

		fVal += fV * fDistCorr;

	}

	projData[angle*projPitch+detector] += fVal;
}

bool FanFP_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float outputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	hipArray* D_dataArray;
	bindVolumeDataTexture(D_volumeData, D_dataArray, volumePitch, dims.iVolWidth, dims.iVolHeight);

	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	dim3 dimBlock(g_detBlockSize, g_anglesPerBlock); // region size, angles
	const unsigned int g_blockSliceSize = g_detBlockSize;

	std::list<hipStream_t> streams;


	unsigned int blockStart = 0;
	unsigned int blockEnd = dims.iProjAngles;

	dim3 dimGrid((blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock,
	             (dims.iProjDets+g_blockSliceSize-1)/g_blockSliceSize); // angle blocks, regions
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	streams.push_back(stream1);
	for (unsigned int i = 0; i < dims.iVolWidth; i += g_blockSlices)
		FanFPhorizontal<<<dimGrid, dimBlock, 0, stream1>>>(D_projData, projPitch, i, blockStart, blockEnd, dims, outputScale);

	hipStream_t stream2;
	hipStreamCreate(&stream2);
	streams.push_back(stream2);
	for (unsigned int i = 0; i < dims.iVolHeight; i += g_blockSlices)
		FanFPvertical<<<dimGrid, dimBlock, 0, stream2>>>(D_projData, projPitch, i, blockStart, blockEnd, dims, outputScale);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipFreeArray(D_dataArray);

	return true;
}

bool FanFP(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float outputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanFP_internal(D_volumeData, volumePitch,
		                         D_projData + iAngle * projPitch, projPitch,
		                         subdims, angles + iAngle,
		                         outputScale);
		if (!ret)
			return false;
	}
	return true;
}

}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_projData;

	SDimensions dims;
	dims.iVolWidth = 128;
	dims.iVolHeight = 128;
	dims.iProjAngles = 180;
	dims.iProjDets = 256;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;
	unsigned int volumePitch, projPitch;

	SFanProjection projs[180];

	projs[0].fSrcX = 0.0f;
	projs[0].fSrcY = 1536.0f;
	projs[0].fDetSX = 128.0f;
	projs[0].fDetSY = -512.0f;
	projs[0].fDetUX = -1.0f;
	projs[0].fDetUY = 0.0f;

#define ROTATE0(name,i,alpha) do { projs[i].f##name##X = projs[0].f##name##X * cos(alpha) - projs[0].f##name##Y * sin(alpha); projs[i].f##name##Y = projs[0].f##name##X * sin(alpha) + projs[0].f##name##Y * cos(alpha); } while(0)

	for (int i = 1; i < 180; ++i) {
		ROTATE0(Src, i, i*2*M_PI/180);
		ROTATE0(DetS, i, i*2*M_PI/180);
		ROTATE0(DetU, i, i*2*M_PI/180);
	}

#undef ROTATE0

	allocateVolume(D_volumeData, dims.iVolWidth, dims.iVolHeight, volumePitch);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_projData, dims.iProjDets, dims.iProjAngles, projPitch);
	printf("pitch: %u\n", projPitch);

	unsigned int y, x;
	float* img = loadImage("phantom128.png", y, x);

	float* sino = new float[dims.iProjAngles * dims.iProjDets];

	memset(sino, 0, dims.iProjAngles * dims.iProjDets * sizeof(float));

	copyVolumeToDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);
	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_projData, projPitch);

	float* angle = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i)
		angle[i] = i*(M_PI/dims.iProjAngles);

	FanFP(D_volumeData, volumePitch, D_projData, projPitch, dims, projs, 1.0f);

	delete[] angle;

	copySinogramFromDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_projData, projPitch);

	float s = 0.0f;
	for (unsigned int y = 0; y < dims.iProjAngles; ++y)
		for (unsigned int x = 0; x < dims.iProjDets; ++x)
			s += sino[y*dims.iProjDets+x] * sino[y*dims.iProjDets+x];
	printf("cpu norm: %f\n", s);

	//zeroVolume(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);
	s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);
	printf("gpu norm: %f\n", s);

	saveImage("sino.png",dims.iProjAngles,dims.iProjDets,sino);


	return 0;
}
#endif
