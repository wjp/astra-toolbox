#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include <cstdio>
#include <cassert>
#include <iostream>


typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_projTexture;


namespace astraCUDA {

const unsigned int g_anglesPerBlock = 16;
const unsigned int g_blockSliceSize = 32;
const unsigned int g_blockSlices = 16;

const unsigned int g_MaxAngles = 2560;

__constant__ float gC_angle_scaled_sin[g_MaxAngles];
__constant__ float gC_angle_scaled_cos[g_MaxAngles];
__constant__ float gC_angle_offset[g_MaxAngles];

static bool bindProjDataTexture(float* data, unsigned int pitch, unsigned int width, unsigned int height, hipTextureAddressMode mode = hipAddressModeBorder)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_projTexture.addressMode[0] = mode;
	gT_projTexture.addressMode[1] = mode;
	gT_projTexture.filterMode = hipFilterModeLinear;
	gT_projTexture.normalized = false;

	hipBindTexture2D(0, gT_projTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);

	// TODO: error value?

	return true;
}

__global__ void devBP(float* D_volData, unsigned int volPitch, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float scaled_cos_theta = gC_angle_scaled_cos[angle];
		const float scaled_sin_theta = gC_angle_scaled_sin[angle];
		const float TOffset = gC_angle_offset[angle];

		const float fT = fX * scaled_cos_theta - fY * scaled_sin_theta + TOffset;
		fVal += tex2D(gT_projTexture, fT, fA);
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

// supersampling version
__global__ void devBP_SS(float* D_volData, unsigned int volPitch, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);

	const float fSubStep = 1.0f/(dims.iRaysPerPixelDim); // * dims.fDetScale);

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	fOutputScale /= (dims.iRaysPerPixelDim * dims.iRaysPerPixelDim);

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float cos_theta = gC_angle_scaled_cos[angle];
		const float sin_theta = gC_angle_scaled_sin[angle];
		const float TOffset = gC_angle_offset[angle];

		float fT = fX * cos_theta - fY * sin_theta + TOffset;

		for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
			float fTy = fT;
			fT += fSubStep * cos_theta;
			for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
				fVal += tex2D(gT_projTexture, fTy, fA);
				fTy -= fSubStep * sin_theta;
			}
		}
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

__global__ void devBP_SART(float* D_volData, unsigned int volPitch, float offset, float angle_sin, float angle_cos, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f );

	const float fT = fX * angle_cos - fY * angle_sin + offset;
	const float fVal = tex2D(gT_projTexture, fT, 0.5f);

	D_volData[Y*volPitch+X] += fVal * fOutputScale;
}


bool BP_internal(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const SParProjection* angles,
        float fOutputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	float* angle_scaled_sin = new float[dims.iProjAngles];
	float* angle_scaled_cos = new float[dims.iProjAngles];
	float* angle_offset = new float[dims.iProjAngles];

	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		double d = angles[i].fDetUX * angles[i].fRayY - angles[i].fDetUY * angles[i].fRayX;
		angle_scaled_cos[i] = angles[i].fRayY / d;
		angle_scaled_sin[i] = -angles[i].fRayX / d; // TODO: Check signs
		angle_offset[i] = (angles[i].fDetSY * angles[i].fRayX - angles[i].fDetSX * angles[i].fRayY) / d;
	}

	hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_scaled_sin), angle_scaled_sin, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e2 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_scaled_cos), angle_scaled_cos, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e3 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_offset), angle_offset, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	assert(e1 == hipSuccess);
	assert(e2 == hipSuccess);
	assert(e3 == hipSuccess);


	delete[] angle_scaled_sin;
	delete[] angle_scaled_cos;
	delete[] angle_offset;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {

		if (dims.iRaysPerPixelDim > 1)
			devBP_SS<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, dims, fOutputScale);
		else
			devBP<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, dims, fOutputScale);
	}
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipStreamDestroy(stream);

	return true;
}

bool BP(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const SParProjection* angles, float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = BP_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle, fOutputScale);
		if (!ret)
			return false;
	}
	return true;
}


bool BP_SART(float* D_volumeData, unsigned int volumePitch,
             float* D_projData, unsigned int projPitch,
             unsigned int angle, const SDimensions& dims,
             const SParProjection* angles, float fOutputScale)
{
	// Only one angle.
	// We need to Clamp to the border pixels instead of to zero, because
	// SART weights with ray length.
	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, 1, hipAddressModeClamp);

	double d = angles[angle].fDetUX * angles[angle].fRayY - angles[angle].fDetUY * angles[angle].fRayX;
	float angle_scaled_cos = angles[angle].fRayY / d;
	float angle_scaled_sin = -angles[angle].fRayX / d; // TODO: Check signs
	float angle_offset = (angles[angle].fDetSY * angles[angle].fRayX - angles[angle].fDetSX * angles[angle].fRayY) / d;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	devBP_SART<<<dimGrid, dimBlock>>>(D_volumeData, volumePitch, angle_offset, angle_scaled_sin, angle_scaled_cos, dims, fOutputScale);
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	return true;
}


}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_projData;

	SDimensions dims;
	dims.iVolWidth = 1024;
	dims.iVolHeight = 1024;
	dims.iProjAngles = 512;
	dims.iProjDets = 1536;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;

	unsigned int volumePitch, projPitch;

	allocateVolume(D_volumeData, dims.iVolWidth, dims.iVolHeight, volumePitch);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_projData, dims.iProjDets, dims.iProjAngles, projPitch);
	printf("pitch: %u\n", projPitch);

	unsigned int y, x;
	float* sino = loadImage("sino.png", y, x);

	float* img = new float[dims.iVolWidth*dims.iVolHeight];

	memset(img, 0, dims.iVolWidth*dims.iVolHeight*sizeof(float));

	copyVolumeToDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);
	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_projData, projPitch);

	float* angle = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i)
		angle[i] = i*(M_PI/dims.iProjAngles);

	BP(D_volumeData, volumePitch, D_projData, projPitch, dims, angle, 0, 1.0f);

	delete[] angle;

	copyVolumeFromDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);

	saveImage("vol.png",dims.iVolHeight,dims.iVolWidth,img);

	return 0;
}
#endif
