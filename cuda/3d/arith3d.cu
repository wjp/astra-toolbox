#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/gpu_runtime_wrapper.h"

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/arith3d.h"

#include "astra/cuda/2d/util.h"

#include <cassert>

namespace astraCUDA3d {

struct opAddScaled {
	__device__ void operator()(float& out, const float in, const float inp) {
		out += in * inp;
	}
};
struct opScaleAndAdd {
	__device__ void operator()(float& out, const float in, const float inp) {
		out = in + out * inp;
	}
};
struct opAddMulScaled {
	__device__ void operator()(float& out, const float in1, const float in2, const float inp) {
		out += in1 * in2 * inp;
	}
};
struct opAddMul {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out += in1 * in2;
	}
};
struct opAdd {
	__device__ void operator()(float& out, const float in) {
		out += in;
	}
};
struct opMul {
	__device__ void operator()(float& out, const float in) {
		out *= in;
	}
};
struct opMul2 {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out *= in1 * in2;
	}
};
struct opDividedBy {
	__device__ void operator()(float& out, const float in) {
		if (out > 0.000001f) // out is assumed to be positive
			out = in / out;
		else
			out = 0.0f;
	}
};
struct opInvert {
	__device__ void operator()(float& out) {
		if (out > 0.000001f) // out is assumed to be positive
			out = 1 / out;
		else
			out = 0.0f;
	}
};
struct opSet {
	__device__ void operator()(float& out, const float inp) {
		out = inp;
	}
};
struct opClampMin {
	__device__ void operator()(float& out, const float inp) {
		if (out < inp)
			out = inp;
	}
};
struct opClampMax {
	__device__ void operator()(float& out, const float inp) {
		if (out > inp)
			out = inp;
	}
};




template<class op, unsigned int repeat>
__global__ void devtoD(float* pfOut, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devFtoD(float* pfOut, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], fParam);
		off += pitch;
		y++;
	}
}


template<class op, unsigned int repeat>
__global__ void devDtoD(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDFtoD(float* pfOut, const float* pfIn, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off], fParam);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDDtoD(float* pfOut, const float* pfIn1, const float* pfIn2, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDDFtoD(float* pfOut, const float* pfIn1, const float* pfIn2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off], fParam);
		off += pitch;
		y++;
	}
}









template<typename op>
bool processVol3D(hipPitchedPtr& out, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processVol3D(hipPitchedPtr& out, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devFtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn = (const float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn = (const float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDFtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn1 = (const float*)in1.ptr;
	const float *pfIn2 = (const float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDDFtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn1, pfIn2, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn1 = (const float*)in1.ptr;
	const float *pfIn2 = (const float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDDtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn1, pfIn2, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}













template<typename op>
bool processSino3D(hipPitchedPtr& out, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processSino3D(hipPitchedPtr& out, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devFtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn = (const float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn = (const float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDFtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn1 = (const float*)in1.ptr;
	const float *pfIn2 = (const float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDDFtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn1, pfIn2, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}

template<typename op>
bool processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims, std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	const float *pfIn1 = (const float*)in1.ptr;
	const float *pfIn2 = (const float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDDtoD<op, 32><<<gridSize, blockSize, 0, stream()>>>(pfOut, pfIn1, pfIn2, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	return stream.syncIfSync(__FUNCTION__);
}


















#define INST_DFtoD(name) \
  template bool processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream); \
  template bool processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream);

#define INST_DtoD(name) \
  template bool processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims, std::optional<hipStream_t> _stream); \
  template bool processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims, std::optional<hipStream_t> _stream);

#define INST_DDtoD(name) \
  template bool processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims, std::optional<hipStream_t> _stream); \
  template bool processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims, std::optional<hipStream_t> _stream);

#define INST_DDFtoD(name) \
  template bool processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream); \
  template bool processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream);


#define INST_toD(name) \
  template bool processVol3D<name>(hipPitchedPtr& out, const SDimensions3D& dims, std::optional<hipStream_t> _stream); \
  template bool processSino3D<name>(hipPitchedPtr& out, const SDimensions3D& dims, std::optional<hipStream_t> _stream);

#define INST_FtoD(name) \
  template bool processVol3D<name>(hipPitchedPtr& out, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream); \
  template bool processSino3D<name>(hipPitchedPtr& out, float fParam, const SDimensions3D& dims, std::optional<hipStream_t> _stream);



INST_DFtoD(opAddScaled)
INST_DFtoD(opScaleAndAdd)
INST_DDFtoD(opAddMulScaled)
INST_DDtoD(opAddMul)
INST_DDtoD(opMul2)
INST_DtoD(opMul)
INST_DtoD(opAdd)
INST_DtoD(opDividedBy)
INST_toD(opInvert)
INST_FtoD(opMul)
INST_FtoD(opSet)
INST_FtoD(opClampMin)
INST_FtoD(opClampMax)


}
