#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/gpu_runtime_wrapper.h"

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA3d {

static const unsigned int g_anglesPerBlock = 4;

// thickness of the slices we're splitting the volume up into
static const unsigned int g_blockSlices = 4;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 1024;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_SrcZ[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetSZ[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];
__constant__ float gC_DetUZ[g_MaxAngles];
__constant__ float gC_DetVX[g_MaxAngles];
__constant__ float gC_DetVY[g_MaxAngles];
__constant__ float gC_DetVZ[g_MaxAngles];


// x=0, y=1, z=2
struct DIR_X {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return x; }
	__device__ float c1(float x, float y, float z) const { return y; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(hipTextureObject_t tex, float f0, float f1, float f2) const { return tex3D<float>(tex, f0, f1, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f0; }
	__device__ float y(float f0, float f1, float f2) const { return f1; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// y=0, x=1, z=2
struct DIR_Y {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return y; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(hipTextureObject_t tex, float f0, float f1, float f2) const { return tex3D<float>(tex, f1, f0, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f0; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// z=0, x=1, y=2
struct DIR_Z {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float c0(float x, float y, float z) const { return z; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return y; }
	__device__ float tex(hipTextureObject_t tex, float f0, float f1, float f2) const { return tex3D<float>(tex, f1, f2, f0); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f2; }
	__device__ float z(float f0, float f1, float f2) const { return f0; }
};

struct SCALE_CUBE {
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1+a2*a2+1.0f) * fOutputScale; }
};

struct SCALE_NONCUBE {
	float fScale1;
	float fScale2;
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1*fScale1+a2*a2*fScale2+1.0f) * fOutputScale; }
};

using TransferConstantsBuffer = TransferConstantsBuffer_t<float>;

bool transferConstants(const SConeProjection* angles, unsigned int iProjAngles,
                       TransferConstantsBuffer& buf, hipStream_t stream)
{
	float* tmp = &(std::get<0>(buf.d))[0];

	// We use an event to assure that the previous transferConstants has completed before
	// re-using the buffer. (Even if it is very unlikely that it hasn't.)
	bool ok = checkCuda(hipStreamWaitEvent(stream, buf.event, 0), "transferConstants wait");

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < iProjAngles; ++i) tmp[i] = angles[i].f##name ; ok &= checkCuda(hipMemcpyToSymbolAsync(HIP_SYMBOL(gC_##name), tmp, iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice, stream), "transferConstants transfer"); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(SrcZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	ok &= checkCuda(hipEventRecord(buf.event, stream), "transferConstants event");

	return ok;
}


	// threadIdx: x = ??? detector  (u?)
	//            y = relative angle

	// blockIdx:  x = ??? detector  (u+v?)
    //            y = angle block

template<class COORD, class SCALE>
__global__ void cone_FP_t(float* D_projData, unsigned int projPitch,
                          hipTextureObject_t tex,
                          unsigned int startSlice,
                          unsigned int startAngle, unsigned int endAngle,
                          const SDimensions3D dims,
                          SCALE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fSrcZ = gC_SrcZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	if (detectorU >= dims.iProjU)
		return;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray from Src to (detectorU,detectorV) from */
		/* X = startSlice to X = endSlice                   */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0) */
		/* ray:   (y) = (ay) * x + (by) */
		/*        (z)   (az)       (bz) */

		const float a1 = (c.c1(fSrcX,fSrcY,fSrcZ) - c.c1(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float a2 = (c.c2(fSrcX,fSrcY,fSrcZ) - c.c2(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float b1 = c.c1(fSrcX,fSrcY,fSrcZ) - a1 * c.c0(fSrcX,fSrcY,fSrcZ);
		const float b2 = c.c2(fSrcX,fSrcY,fSrcZ) - a2 * c.c0(fSrcX,fSrcY,fSrcZ);

		const float fDistCorr = sc.scale(a1, a2);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(tex, f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr;

		D_projData[(size_t)(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

template<class COORD>
__global__ void cone_FP_SS_t(float* D_projData, unsigned int projPitch,
                             hipTextureObject_t tex,
                             unsigned int startSlice,
                             unsigned int startAngle, unsigned int endAngle,
                             const SDimensions3D dims, int iRaysPerDetDim,
                             SCALE_NONCUBE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fSrcZ = gC_SrcZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	if (detectorU >= dims.iProjU)
		return;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	const float fSubStep = 1.0f/iRaysPerDetDim;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray from Src to (detectorU,detectorV) from */
		/* X = startSlice to X = endSlice                   */

		float fV = 0.0f;

		float fdU = detectorU - 0.5f + 0.5f*fSubStep;
		for (int iSubU = 0; iSubU < iRaysPerDetDim; ++iSubU, fdU+=fSubStep) {
		float fdV = detectorV - 0.5f + 0.5f*fSubStep;
		for (int iSubV = 0; iSubV < iRaysPerDetDim; ++iSubV, fdV+=fSubStep) {

		const float fDetX = fDetSX + fdU*fDetUX + fdV*fDetVX;
		const float fDetY = fDetSY + fdU*fDetUY + fdV*fDetVY;
		const float fDetZ = fDetSZ + fdU*fDetUZ + fdV*fDetVZ;

		/*        (x)   ( 1)       ( 0) */
		/* ray:   (y) = (ay) * x + (by) */
		/*        (z)   (az)       (bz) */

		const float a1 = (c.c1(fSrcX,fSrcY,fSrcZ) - c.c1(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float a2 = (c.c2(fSrcX,fSrcY,fSrcZ) - c.c2(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float b1 = c.c1(fSrcX,fSrcY,fSrcZ) - a1 * c.c0(fSrcX,fSrcY,fSrcZ);
		const float b2 = c.c2(fSrcX,fSrcY,fSrcZ) - a2 * c.c0(fSrcX,fSrcY,fSrcZ);

		const float fDistCorr = sc.scale(a1, a2);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(tex, f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr;
		fV += fVal;

		}
		}

		D_projData[(size_t)(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fV / (iRaysPerDetDim * iRaysPerDetDim);
	}
}


bool ConeFP_Array_internal(hipPitchedPtr D_projData,
                  hipTextureObject_t D_texObj,
                  const SDimensions3D& dims,
                  unsigned int angleCount, const SConeProjection* angles,
                  const SProjectorParams3D& params, hipStream_t stream)
{
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	bool cube = true;
	if (abs(params.volScale.fX / params.volScale.fY - 1.0) > 0.00001)
		cube = false;
	if (abs(params.volScale.fX / params.volScale.fZ - 1.0) > 0.00001)
		cube = false;

	SCALE_CUBE scube;
	scube.fOutputScale = params.fOutputScale * params.volScale.fX;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.volScale.fY / params.volScale.fX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.volScale.fZ / params.volScale.fX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.volScale.fX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.volScale.fX / params.volScale.fY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.volScale.fZ / params.volScale.fY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.volScale.fY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.volScale.fX / params.volScale.fZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.volScale.fY / params.volScale.fZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.volScale.fZ;

	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= angleCount; ++a) {
		int dir = -1;
		if (a != angleCount) {
			float dX = fabsf(angles[a].fSrcX - (angles[a].fDetSX + dims.iProjU*angles[a].fDetUX*0.5f + dims.iProjV*angles[a].fDetVX*0.5f));
			float dY = fabsf(angles[a].fSrcY - (angles[a].fDetSY + dims.iProjU*angles[a].fDetUY*0.5f + dims.iProjV*angles[a].fDetVY*0.5f));
			float dZ = fabsf(angles[a].fSrcZ - (angles[a].fDetSZ + dims.iProjU*angles[a].fDetUZ*0.5f + dims.iProjV*angles[a].fDetVZ*0.5f));

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == angleCount || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							if (cube)
								cone_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, scube);
							else
								cone_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, snoncubeX);
						else
							cone_FP_SS_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeX);
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							if (cube)
								cone_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, scube);
							else
								cone_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, snoncubeY);
						else
							cone_FP_SS_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeY);
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							if (cube)
								cone_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, scube);
							else
								cone_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, snoncubeZ);
						else
							cone_FP_SS_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), D_texObj, i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeZ);
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	// printf("%f\n", toc(t));

	return true;
}


bool ConeFP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	TransferConstantsBuffer tcbuf(g_MaxAngles);

	hipStream_t stream;
	if (!checkCuda(hipStreamCreate(&stream), "ConeFP stream"))
		return false;

	// transfer volume to array
	hipArray* cuArray = allocateVolumeArray(dims);
	if (!cuArray) {
		hipStreamDestroy(stream);
		return false;
	}

	hipTextureObject_t D_texObj;
	if (!createTextureObject3D(cuArray, D_texObj)) {
		hipStreamDestroy(stream);
		hipFreeArray(cuArray);
		return false;
	}

	if (!transferVolumeToArray(D_volumeData, cuArray, dims, stream)) {
		hipDestroyTextureObject(D_texObj);
		hipStreamDestroy(stream);
		hipFreeArray(cuArray);
		return false;
	}

	bool ok = true;

	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;

		ok = transferConstants(angles + iAngle, iEndAngle - iAngle, tcbuf, stream);
		if (!ok)
			break;

		hipPitchedPtr D_subprojData = D_projData;
		D_subprojData.ptr = (char*)D_projData.ptr + iAngle * D_projData.pitch;

		ok = ConeFP_Array_internal(D_subprojData, D_texObj,
		                            dims, iEndAngle - iAngle, angles + iAngle,
		                            params, stream);
		if (!ok)
			break;
	}

	ok &= checkCuda(hipStreamSynchronize(stream), "ConeFP sync");

	hipDestroyTextureObject(D_texObj);
	hipFreeArray(cuArray);
	hipStreamDestroy(stream);

	return ok;
}


}
