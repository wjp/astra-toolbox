#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2021, imec Vision Lab, University of Antwerp
           2014-2021, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "astra/cuda/3d/util3d.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include "astra/cuda/3d/dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneLineVolumeTexture;
static texture3D gT_coneLineProjTexture;


namespace astraCUDA3d {

static const unsigned int g_anglesPerBlock = 4;

// thickness of the slices we're splitting the volume up into
static const unsigned int g_blockSlices = 16;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 16;

static const unsigned g_MaxAngles = 1024;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_SrcZ[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetSZ[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];
__constant__ float gC_DetUZ[g_MaxAngles];
__constant__ float gC_DetVX[g_MaxAngles];
__constant__ float gC_DetVY[g_MaxAngles];
__constant__ float gC_DetVZ[g_MaxAngles];



static bool bindVolumeDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneLineVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneLineVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneLineVolumeTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneLineVolumeTexture.filterMode = hipFilterModeLinear;
	gT_coneLineVolumeTexture.normalized = false;

	hipBindTextureToArray(gT_coneLineVolumeTexture, array, channelDesc);

	// TODO: error value?

	return true;
}

static bool bindProjDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneLineProjTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneLineProjTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneLineProjTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneLineProjTexture.filterMode = hipFilterModeLinear;
	gT_coneLineProjTexture.normalized = false;

	hipBindTextureToArray(gT_coneLineProjTexture, array, channelDesc);

	// TODO: error value?

	return true;
}



// x=0, y=1, z=2
struct DIR_X {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return x; }
	__device__ float c1(float x, float y, float z) const { return y; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_coneLineVolumeTexture, f0, f1, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f0; }
	__device__ float y(float f0, float f1, float f2) const { return f1; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
	__device__ int ix(int f0, int f1, int f2) const { return f0; }
	__device__ int iy(int f0, int f1, int f2) const { return f1; }
	__device__ int iz(int f0, int f1, int f2) const { return f2; }
};

// y=0, x=1, z=2
struct DIR_Y {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return y; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_coneLineVolumeTexture, f1, f0, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f0; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
	__device__ int ix(int f0, int f1, int f2) const { return f1; }
	__device__ int iy(int f0, int f1, int f2) const { return f0; }
	__device__ int iz(int f0, int f1, int f2) const { return f2; }
};

// z=0, x=1, y=2
struct DIR_Z {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float c0(float x, float y, float z) const { return z; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return y; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_coneLineVolumeTexture, f1, f2, f0); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f2; }
	__device__ float z(float f0, float f1, float f2) const { return f0; }
	__device__ int ix(int f0, int f1, int f2) const { return f1; }
	__device__ int iy(int f0, int f1, int f2) const { return f2; }
	__device__ int iz(int f0, int f1, int f2) const { return f0; }
};

struct SCALE_CUBE {
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1*a2*a2+1.0f) * fOutputScale; }
};

struct SCALE_NONCUBE {
	float fScale1;
	float fScale2;
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1*fScale1+a2*a2*fScale2+1.0f) * fOutputScale; }
};




template<class COORD, class SCALE>
__global__ void cone_FP_line_t(float* D_projData, unsigned int projPitch,
                          unsigned int startSlice,
                          unsigned int startAngle, unsigned int endAngle,
                          const SDimensions3D dims, SCALE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fSrcZ = gC_SrcZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	if (detectorU >= dims.iProjU)
		return;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray from Src to (detectorU,detectorV) from */
		/* X = startSlice to X = endSlice                   */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0) */
		/* ray:   (y) = (ay) * T + (by) */
		/*        (z)   (az)       (bz) */

		float a1 = (c.c1(fSrcX,fSrcY,fSrcZ) - c.c1(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		float a2 = (c.c2(fSrcX,fSrcY,fSrcZ) - c.c2(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float b1 = c.c1(fSrcX,fSrcY,fSrcZ) - a1 * c.c0(fSrcX,fSrcY,fSrcZ);
		const float b2 = c.c2(fSrcX,fSrcY,fSrcZ) - a2 * c.c0(fSrcX,fSrcY,fSrcZ);

		const float fDistCorr = sc.scale(a1, a2);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;

		float fCurT = startSlice - 0.5f*c.nSlices(dims);
		// Coordinates
		float f1 = a1 * fCurT + b1;
		float f2 = a2 * fCurT + b2;
		// Fractional index
		f1 += 0.5f*c.nDim1(dims);
		f2 += 0.5f*c.nDim2(dims);
		// Detector index
		f1 = floor(f1);
		f2 = floor(f2);

		float fNext0 = fCurT + 1.0f;

		float fNext1 = f1;
		if (a1 > 0) fNext1 += 1.0f;
		fNext1 = (fNext1 - 0.5f*c.nDim1(dims) - b1) / a1;

		float fNext2 = f2;
		if (a2 > 0) fNext2 += 1.0f;
		fNext2 = (fNext2 - 0.5f*c.nDim2(dims) - b2) / a2;

		// Texture coordinates (at center of detector)
		f1 += 0.5f;
		f2 += 0.5f;

		if (a1 == 0.0f) fNext1 = 1.0f/0.0f;
		if (a2 == 0.0f) fNext2 = 1.0f/0.0f;

		float fStep1 = 1.0f;
		if (a1 < 0) fStep1 = -1.0f;
		float fStep2 = 1.0f;
		if (a2 < 0) fStep2 = -1.0f;
		a1 = fabsf(1.0f / a1);
		a2 = fabsf(1.0f / a2);


		//if (fNext0 < fCurT) { printf("0: %f %f\n", fNext0, fCurT); return; }
		//if (fNext1 < fCurT) { printf("0: %f %f\n", fNext1, fCurT); return; }
		//if (fNext2 < fCurT) { printf("0: %f %f\n", fNext2, fCurT); return; }

		while (fNext0 <= endSlice - 0.5f*c.nSlices(dims))
		{
			float fV = c.tex(f0, f1, f2);
			float fPrevT = fCurT;
			if (fNext0 <= fNext1 && fNext0 <= fNext2) {
				// Step in X dir
				//if (fNext0 >= endSlice - 0.5f*c.nSlices(dims))
				//	break;
				fCurT = fNext0;
				fNext0 += 1.0f;
				f0 += 1.0f;
			} else if (fNext1 <= fNext0 && fNext1 <= fNext2) {
				// Step in Y dir
				fCurT = fNext1;
				fNext1 += a1;
				f1 += fStep1;
			} else {
				// Step in Z dir
				fCurT = fNext2;
				fNext2 += a2;
				f2 += fStep2;
			}
			fVal += (fCurT - fPrevT) * fV;
			//fVal += fV;
		}

		fVal *= fDistCorr;

		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

template<class COORD, class SCALE>
__global__ void cone_BP_line_t(float* D_volData, unsigned int volPitch,
                          unsigned int startSlice,
                          unsigned int startAngle, unsigned int endAngle,
                          unsigned int angleOffset,
                          const SDimensions3D dims, SCALE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fSrcZ = gC_SrcZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	if (detectorU >= dims.iProjU)
		return;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray from Src to (detectorU,detectorV) from */
		/* X = startSlice to X = endSlice                   */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0) */
		/* ray:   (y) = (ay) * T + (by) */
		/*        (z)   (az)       (bz) */

		float a1 = (c.c1(fSrcX,fSrcY,fSrcZ) - c.c1(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		float a2 = (c.c2(fSrcX,fSrcY,fSrcZ) - c.c2(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float b1 = c.c1(fSrcX,fSrcY,fSrcZ) - a1 * c.c0(fSrcX,fSrcY,fSrcZ);
		const float b2 = c.c2(fSrcX,fSrcY,fSrcZ) - a2 * c.c0(fSrcX,fSrcY,fSrcZ);

		const float fDistCorr = sc.scale(a1, a2);

		float fCurT = startSlice - 0.5f*c.nSlices(dims);
		// Coordinates
		float f1 = a1 * fCurT + b1;
		float f2 = a2 * fCurT + b2;
		// Fractional index
		f1 += 0.5f*c.nDim1(dims);
		f2 += 0.5f*c.nDim2(dims);
		// Voxel index
		f1 = floor(f1);
		f2 = floor(f2);

		float fNext0 = fCurT + 1.0f;

		float fNext1 = f1;
		if (a1 > 0) fNext1 += 1.0f;
		fNext1 = (fNext1 - 0.5f*c.nDim1(dims) - b1) / a1;

		float fNext2 = f2;
		if (a2 > 0) fNext2 += 1.0f;
		fNext2 = (fNext2 - 0.5f*c.nDim2(dims) - b2) / a2;

		// Indices
		int c0 = startSlice;
		int c1 = (int)f1; // already rounded
		int c2 = (int)f2; // already rounded

		int iStep1 = 1;
		if (a1 < 0) iStep1 = -1;
		int iStep2 = 1;
		if (a2 < 0) iStep2 = -1;

		if (a1 == 0.0f) fNext1 = 1.0f/0.0f;
		if (a2 == 0.0f) fNext2 = 1.0f/0.0f;

		a1 = fabsf(1.0f / a1);
		a2 = fabsf(1.0f / a2);

		float fVal = tex3D(gT_coneLineProjTexture, detectorU + 0.5f, angle + angleOffset + 0.5f, detectorV + 0.5f) * fDistCorr;

		while (c0 < endSlice)
		{
			float* addr = 0;
			if (c1 >= 0 && c1 < c.nDim1(dims) && c2 >= 0 && c2 < c.nDim2(dims)) {
				addr = &D_volData[(c.iz(c0,c1,c2)*dims.iVolY +c.iy(c0,c1,c2))*volPitch + c.ix(c0,c1,c2)];
			}
			float fPrevT = fCurT;
			if (fNext0 <= fNext1 && fNext0 <= fNext2) {
				// Step in X dir
				fCurT = fNext0;
				fNext0 += 1.0f;
				c0 += 1;
			} else if (fNext1 <= fNext0 && fNext1 <= fNext2) {
				// Step in Y dir
				fCurT = fNext1;
				fNext1 += a1;
				c1 += iStep1;
			} else {
				// Step in Z dir
				fCurT = fNext2;
				fNext2 += a2;
				c2 += iStep2;
			}
			if (addr) {
				atomicAdd(addr, (fCurT - fPrevT) * fVal );
			}
		}
	}
}



bool ConeLineFP_Array_internal(hipPitchedPtr D_projData,
                  const SDimensions3D& dims, unsigned int angleCount, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	// transfer angles to constant memory
	float* tmp = new float[angleCount];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, angleCount*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(SrcZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	bool cube = true;
	if (abs(params.fVolScaleX / params.fVolScaleY - 1.0) > 0.00001)
		cube = false;
	if (abs(params.fVolScaleX / params.fVolScaleZ - 1.0) > 0.00001)
		cube = false;

	SCALE_CUBE scube;
	scube.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;


	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= angleCount; ++a) {
		int dir = -1;
		if (a != angleCount) {
			float dX = fabsf(angles[a].fSrcX - (angles[a].fDetSX + dims.iProjU*angles[a].fDetUX*0.5f + dims.iProjV*angles[a].fDetVX*0.5f));
			float dY = fabsf(angles[a].fSrcY - (angles[a].fDetSY + dims.iProjU*angles[a].fDetUY*0.5f + dims.iProjV*angles[a].fDetVY*0.5f));
			float dZ = fabsf(angles[a].fSrcZ - (angles[a].fDetSZ + dims.iProjU*angles[a].fDetUZ*0.5f + dims.iProjV*angles[a].fDetVZ*0.5f));

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == angleCount || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (cube)
							cone_FP_line_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
						else
							cone_FP_line_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeX);
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (cube)
							cone_FP_line_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
						else
							cone_FP_line_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeY);
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (cube)
							cone_FP_line_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
						else
							cone_FP_line_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeZ);
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	bool ok = true;

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter) {
		ok &= checkCuda(hipStreamSynchronize(*iter), "cone line fp");
		hipStreamDestroy(*iter);
	}

	// printf("%f\n", toc(t));

	return ok;
}

bool ConeLineBP_Array_internal(hipPitchedPtr D_volData,
                  const SDimensions3D& dims, unsigned int startAngle, unsigned int angleCount, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	angles += startAngle;

	// transfer angles to constant memory
	float* tmp = new float[angleCount];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, angleCount*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(SrcZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	bool cube = true;
	if (abs(params.fVolScaleX / params.fVolScaleY - 1.0) > 0.00001)
		cube = false;
	if (abs(params.fVolScaleX / params.fVolScaleZ - 1.0) > 0.00001)
		cube = false;

	SCALE_CUBE scube;
	scube.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;


	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= angleCount; ++a) {
		int dir = -1;
		if (a != angleCount) {
			float dX = fabsf(angles[a].fSrcX - (angles[a].fDetSX + dims.iProjU*angles[a].fDetUX*0.5f + dims.iProjV*angles[a].fDetVX*0.5f));
			float dY = fabsf(angles[a].fSrcY - (angles[a].fDetSY + dims.iProjU*angles[a].fDetUY*0.5f + dims.iProjV*angles[a].fDetVY*0.5f));
			float dZ = fabsf(angles[a].fSrcZ - (angles[a].fDetSZ + dims.iProjU*angles[a].fDetUZ*0.5f + dims.iProjV*angles[a].fDetVZ*0.5f));

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == angleCount || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (cube)
							cone_BP_line_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_volData.ptr, D_volData.pitch/sizeof(float), i, blockStart, blockEnd, startAngle, dims, scube);
						else
							cone_BP_line_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_volData.ptr, D_volData.pitch/sizeof(float), i, blockStart, blockEnd, startAngle, dims, snoncubeX);
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (cube)
							cone_BP_line_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_volData.ptr, D_volData.pitch/sizeof(float), i, blockStart, blockEnd, startAngle, dims, scube);
						else
							cone_BP_line_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_volData.ptr, D_volData.pitch/sizeof(float), i, blockStart, blockEnd, startAngle, dims, snoncubeY);
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (cube)
							cone_BP_line_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_volData.ptr, D_volData.pitch/sizeof(float), i, blockStart, blockEnd, startAngle, dims, scube);
						else
							cone_BP_line_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_volData.ptr, D_volData.pitch/sizeof(float), i, blockStart, blockEnd, startAngle, dims, snoncubeZ);
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	bool ok = true;

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter) {
		ok &= checkCuda(hipStreamSynchronize(*iter), "cone line bp");
		hipStreamDestroy(*iter);
	}

	// printf("%f\n", toc(t));

	return ok;
}


bool ConeLineFP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer volume to array

	hipArray* cuArray = allocateVolumeArray(dims);
	transferVolumeToArray(D_volumeData, cuArray, dims);
	bindVolumeDataTexture(cuArray);

	bool ret;

	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;

		hipPitchedPtr D_subprojData = D_projData;
		D_subprojData.ptr = (char*)D_projData.ptr + iAngle * D_projData.pitch;

		ret = ConeLineFP_Array_internal(D_subprojData,
		                            dims, iEndAngle - iAngle, angles + iAngle,
		                            params);
		if (!ret)
			break;
	}

	hipFreeArray(cuArray);

	return ret;
}

bool ConeLineBP_Array(hipPitchedPtr D_volumeData,
                  hipArray *D_projArray,
                  const SDimensions3D& dims, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	bindProjDataTexture(D_projArray);

	bool ret = true;

	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;

		ret = ConeLineBP_Array_internal(D_volumeData, dims, iAngle, iEndAngle - iAngle, angles, params);
		if (!ret)
			break;
	}

	return ret;
}

bool ConeLineBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = ConeLineBP_Array(D_volumeData, cuArray, dims, angles, params);

	hipFreeArray(cuArray);

	return ret;
}



}
