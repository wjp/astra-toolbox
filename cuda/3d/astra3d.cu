/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/gpu_runtime_wrapper.h"

#include "astra/cuda/3d/cgls3d.h"
#include "astra/cuda/3d/sirt3d.h"
#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/cone_fp.h"
#include "astra/cuda/3d/cone_bp.h"
#include "astra/cuda/3d/par3d_fp.h"
#include "astra/cuda/3d/par3d_bp.h"
#include "astra/cuda/3d/fdk.h"
#include "astra/cuda/3d/arith3d.h"
#include "astra/cuda/3d/astra3d.h"
#include "astra/cuda/3d/mem3d.h"

#include "astra/Data3D.h"
#include "astra/Logging.h"

#include "astra/GeometryUtil3D.h"

#include <iostream>
#include <cstdio>
#include <cassert>

using namespace astraCUDA3d;

namespace astra {

class AstraSIRT3d_internal {
public:
	SDimensions3D dims;
	SProjectorParams3D params;

	float* angles;
	float fOriginSourceDistance;
	float fOriginDetectorDistance;
	float fRelaxation;

	Geometry3DParameters projs;

	bool initialized;
	bool setStartReconstruction;

	bool useVolumeMask;
	bool useSinogramMask;

	// Input/output
	hipPitchedPtr D_projData;
	hipPitchedPtr D_volumeData;
	hipPitchedPtr D_maskData;
	hipPitchedPtr D_smaskData;

	SIRT sirt;
};

AstraSIRT3d::AstraSIRT3d()
{
	pData = new AstraSIRT3d_internal();

	pData->angles = 0;
	pData->D_projData.ptr = 0;
	pData->D_volumeData.ptr = 0;
	pData->D_maskData.ptr = 0;
	pData->D_smaskData.ptr = 0;

	pData->dims.iVolX = 0;
	pData->dims.iVolY = 0;
	pData->dims.iVolZ = 0;
	pData->dims.iProjAngles = 0;
	pData->dims.iProjU = 0;
	pData->dims.iProjV = 0;

	pData->fRelaxation = 1.0f;

	pData->initialized = false;
	pData->setStartReconstruction = false;

	pData->useVolumeMask = false;
	pData->useSinogramMask = false;
}

AstraSIRT3d::~AstraSIRT3d()
{
	delete[] pData->angles;
	pData->angles = 0;

	hipFree(pData->D_projData.ptr);
	pData->D_projData.ptr = 0;

	hipFree(pData->D_volumeData.ptr);
	pData->D_volumeData.ptr = 0;

	hipFree(pData->D_maskData.ptr);
	pData->D_maskData.ptr = 0;

	hipFree(pData->D_smaskData.ptr);
	pData->D_smaskData.ptr = 0;

	delete pData;
	pData = 0;
}

bool AstraSIRT3d::setGeometry(const CVolumeGeometry3D* pVolGeom,
	                      const CProjectionGeometry3D* pProjGeom)
{
	if (pData->initialized)
		return false;

	bool ok = astra::convertAstraGeometry_dims(pVolGeom, pProjGeom, pData->dims);

	if (!ok)
		return false;

	pData->projs = astra::convertAstraGeometry(pVolGeom, pProjGeom, pData->params.volScale);

	if (!pData->projs.isValid())
		return false;

	return true;
}


bool AstraSIRT3d::enableSuperSampling(unsigned int iVoxelSuperSampling,
                                      unsigned int iDetectorSuperSampling)
{
	if (pData->initialized)
		return false;

	if (iVoxelSuperSampling == 0 || iDetectorSuperSampling == 0)
		return false;

	pData->params.iRaysPerVoxelDim = iVoxelSuperSampling;
	pData->params.iRaysPerDetDim = iDetectorSuperSampling;

	return true;
}

bool AstraSIRT3d::setRelaxation(float r)
{
	if (pData->initialized)
		return false;

	pData->fRelaxation = r;

	return true;
}

bool AstraSIRT3d::enableVolumeMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->sirt.enableVolumeMask();
	pData->useVolumeMask = ok;

	return ok;
}

bool AstraSIRT3d::enableSinogramMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->sirt.enableSinogramMask();
	pData->useSinogramMask = ok;

	return ok;
}
	
bool AstraSIRT3d::setGPUIndex(int index)
{
	if (index != -1) {
		hipSetDevice(index);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	return true;
}

bool AstraSIRT3d::init()
{
	if (pData->initialized)
		return false;

	if (pData->dims.iVolX == 0 || pData->dims.iProjAngles == 0)
		return false;

	bool ok;

	if (pData->projs.isParallel())
		ok = pData->sirt.setPar3DGeometry(pData->dims, pData->projs.getParallel(), pData->params);
	else if (pData->projs.isCone())
		ok = pData->sirt.setConeGeometry(pData->dims, pData->projs.getCone(), pData->params);
	else
		ok = false;

	if (!ok)
		return false;

	pData->sirt.setRelaxation(pData->fRelaxation);

	ok = pData->sirt.init();
	if (!ok)
		return false;

	pData->D_volumeData = allocateVolumeData(pData->dims);
	ok = pData->D_volumeData.ptr;
	if (!ok)
		return false;

	pData->D_projData = allocateProjectionData(pData->dims);
	ok = pData->D_projData.ptr;
	if (!ok) {
		hipFree(pData->D_volumeData.ptr);
		pData->D_volumeData.ptr = 0;
		return false;
	}

	if (pData->useVolumeMask) {
		pData->D_maskData = allocateVolumeData(pData->dims);
		ok = pData->D_maskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			return false;
		}
	}

	if (pData->useSinogramMask) {
		pData->D_smaskData = allocateProjectionData(pData->dims);
		ok = pData->D_smaskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			hipFree(pData->D_maskData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			pData->D_maskData.ptr = 0;
			return false;
		}
	}

	pData->initialized = true;

	return true;
}

bool AstraSIRT3d::setMinConstraint(float fMin)
{
	if (!pData->initialized)
		return false;
	return pData->sirt.setMinConstraint(fMin);
}

bool AstraSIRT3d::setMaxConstraint(float fMax)
{
	if (!pData->initialized)
		return false;
	return pData->sirt.setMaxConstraint(fMax);
}

bool AstraSIRT3d::setSinogram(const float* pfSinogram,
                              unsigned int iSinogramPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfSinogram)
		return false;

	bool ok = copyProjectionsToDevice(pfSinogram, pData->D_projData, pData->dims, iSinogramPitch);

	if (!ok)
		return false;

	ok = pData->sirt.setBuffers(pData->D_volumeData, pData->D_projData);
	if (!ok)
		return false;

	pData->setStartReconstruction = false;

	return true;
}

bool AstraSIRT3d::setVolumeMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useVolumeMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyVolumeToDevice(pfMask, pData->D_maskData,
	                             pData->dims, iMaskPitch);
	if (!ok)
		return false;

	ok = pData->sirt.setVolumeMask(pData->D_maskData);
	if (!ok)
		return false;

	return true;
}

bool AstraSIRT3d::setSinogramMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useSinogramMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyProjectionsToDevice(pfMask, pData->D_smaskData, pData->dims, iMaskPitch);

	if (!ok)
		return false;

	ok = pData->sirt.setSinogramMask(pData->D_smaskData);
	if (!ok)
		return false;

	return true;
}

bool AstraSIRT3d::setStartReconstruction(const float* pfReconstruction,
                                         unsigned int iReconstructionPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfReconstruction)
		return false;

	bool ok = copyVolumeToDevice(pfReconstruction, pData->D_volumeData,
	                             pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	pData->setStartReconstruction = true;

	return true;
}

bool AstraSIRT3d::iterate(unsigned int iIterations)
{
	if (!pData->initialized)
		return false;

	if (!pData->setStartReconstruction)
		zeroVolumeData(pData->D_volumeData, pData->dims);

	bool ok = pData->sirt.iterate(iIterations);
	if (!ok)
		return false;

	return true;
}

bool AstraSIRT3d::getReconstruction(float* pfReconstruction,
                                    unsigned int iReconstructionPitch) const
{
	if (!pData->initialized)
		return false;

	bool ok = copyVolumeFromDevice(pfReconstruction, pData->D_volumeData,
	                               pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	return true;
}

float AstraSIRT3d::computeDiffNorm()
{
	if (!pData->initialized)
		return 0.0f; // FIXME: Error?

	return pData->sirt.computeDiffNorm();
}




class AstraCGLS3d_internal {
public:
	SDimensions3D dims;
	SProjectorParams3D params;

	float* angles;
	float fOriginSourceDistance;
	float fOriginDetectorDistance;

	Geometry3DParameters projs;

	bool initialized;
	bool setStartReconstruction;

	bool useVolumeMask;
	bool useSinogramMask;

	// Input/output
	hipPitchedPtr D_projData;
	hipPitchedPtr D_volumeData;
	hipPitchedPtr D_maskData;
	hipPitchedPtr D_smaskData;

	CGLS cgls;
};

AstraCGLS3d::AstraCGLS3d()
{
	pData = new AstraCGLS3d_internal();

	pData->angles = 0;
	pData->D_projData.ptr = 0;
	pData->D_volumeData.ptr = 0;
	pData->D_maskData.ptr = 0;
	pData->D_smaskData.ptr = 0;

	pData->dims.iVolX = 0;
	pData->dims.iVolY = 0;
	pData->dims.iVolZ = 0;
	pData->dims.iProjAngles = 0;
	pData->dims.iProjU = 0;
	pData->dims.iProjV = 0;

	pData->initialized = false;
	pData->setStartReconstruction = false;

	pData->useVolumeMask = false;
	pData->useSinogramMask = false;
}

AstraCGLS3d::~AstraCGLS3d()
{
	delete[] pData->angles;
	pData->angles = 0;

	hipFree(pData->D_projData.ptr);
	pData->D_projData.ptr = 0;

	hipFree(pData->D_volumeData.ptr);
	pData->D_volumeData.ptr = 0;

	hipFree(pData->D_maskData.ptr);
	pData->D_maskData.ptr = 0;

	hipFree(pData->D_smaskData.ptr);
	pData->D_smaskData.ptr = 0;

	delete pData;
	pData = 0;
}

bool AstraCGLS3d::setGeometry(const CVolumeGeometry3D* pVolGeom,
	                      const CProjectionGeometry3D* pProjGeom)
{
	if (pData->initialized)
		return false;

	bool ok = astra::convertAstraGeometry_dims(pVolGeom, pProjGeom, pData->dims);

	if (!ok)
		return false;

	pData->projs = astra::convertAstraGeometry(pVolGeom, pProjGeom, pData->params.volScale);

	if (!pData->projs.isValid())
		return false;

	return true;
}

bool AstraCGLS3d::enableSuperSampling(unsigned int iVoxelSuperSampling,
                                      unsigned int iDetectorSuperSampling)
{
	if (pData->initialized)
		return false;

	if (iVoxelSuperSampling == 0 || iDetectorSuperSampling == 0)
		return false;

	pData->params.iRaysPerVoxelDim = iVoxelSuperSampling;
	pData->params.iRaysPerDetDim = iDetectorSuperSampling;

	return true;
}

bool AstraCGLS3d::enableVolumeMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->cgls.enableVolumeMask();
	pData->useVolumeMask = ok;

	return ok;
}

#if 0
bool AstraCGLS3d::enableSinogramMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->cgls.enableSinogramMask();
	pData->useSinogramMask = ok;

	return ok;
}
#endif
	
bool AstraCGLS3d::setGPUIndex(int index)
{
	if (index != -1) {
		hipSetDevice(index);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	return true;
}

bool AstraCGLS3d::init()
{
	if (pData->initialized)
		return false;

	if (pData->dims.iVolX == 0 || pData->dims.iProjAngles == 0)
		return false;

	bool ok;

	if (pData->projs.isParallel())
		ok = pData->cgls.setPar3DGeometry(pData->dims, pData->projs.getParallel(), pData->params);
	else if (pData->projs.isCone())
		ok = pData->cgls.setConeGeometry(pData->dims, pData->projs.getCone(), pData->params);
	else
		ok = false;

	if (!ok)
		return false;

	ok = pData->cgls.init();
	if (!ok)
		return false;

	pData->D_volumeData = allocateVolumeData(pData->dims);
	ok = pData->D_volumeData.ptr;
	if (!ok)
		return false;

	pData->D_projData = allocateProjectionData(pData->dims);
	ok = pData->D_projData.ptr;
	if (!ok) {
		hipFree(pData->D_volumeData.ptr);
		pData->D_volumeData.ptr = 0;
		return false;
	}

	if (pData->useVolumeMask) {
		pData->D_maskData = allocateVolumeData(pData->dims);
		ok = pData->D_maskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			return false;
		}
	}

	if (pData->useSinogramMask) {
		pData->D_smaskData = allocateProjectionData(pData->dims);
		ok = pData->D_smaskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			hipFree(pData->D_maskData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			pData->D_maskData.ptr = 0;
			return false;
		}
	}

	pData->initialized = true;

	return true;
}

#if 0
bool AstraCGLS3d::setMinConstraint(float fMin)
{
	if (!pData->initialized)
		return false;
	return pData->cgls.setMinConstraint(fMin);
}

bool AstraCGLS3d::setMaxConstraint(float fMax)
{
	if (!pData->initialized)
		return false;
	return pData->cgls.setMaxConstraint(fMax);
}
#endif

bool AstraCGLS3d::setSinogram(const float* pfSinogram,
                              unsigned int iSinogramPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfSinogram)
		return false;

	bool ok = copyProjectionsToDevice(pfSinogram, pData->D_projData, pData->dims, iSinogramPitch);

	if (!ok)
		return false;

	ok = pData->cgls.setBuffers(pData->D_volumeData, pData->D_projData);
	if (!ok)
		return false;

	pData->setStartReconstruction = false;

	return true;
}

bool AstraCGLS3d::setVolumeMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useVolumeMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyVolumeToDevice(pfMask, pData->D_maskData,
	                             pData->dims, iMaskPitch);
	if (!ok)
		return false;

	ok = pData->cgls.setVolumeMask(pData->D_maskData);
	if (!ok)
		return false;

	return true;
}

#if 0
bool AstraCGLS3d::setSinogramMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useSinogramMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyProjectionsToDevice(pfMask, pData->D_smaskData, pData->dims, iMaskPitch);

	if (!ok)
		return false;

	ok = pData->cgls.setSinogramMask(pData->D_smaskData);
	if (!ok)
		return false;

	return true;
}
#endif

bool AstraCGLS3d::setStartReconstruction(const float* pfReconstruction,
                                         unsigned int iReconstructionPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfReconstruction)
		return false;

	bool ok = copyVolumeToDevice(pfReconstruction, pData->D_volumeData,
	                             pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	pData->setStartReconstruction = true;

	return true;
}

bool AstraCGLS3d::iterate(unsigned int iIterations)
{
	if (!pData->initialized)
		return false;

	if (!pData->setStartReconstruction)
		zeroVolumeData(pData->D_volumeData, pData->dims);

	bool ok = pData->cgls.iterate(iIterations);
	if (!ok)
		return false;

	return true;
}

bool AstraCGLS3d::getReconstruction(float* pfReconstruction,
                                    unsigned int iReconstructionPitch) const
{
	if (!pData->initialized)
		return false;

	bool ok = copyVolumeFromDevice(pfReconstruction, pData->D_volumeData,
	                               pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	return true;
}

float AstraCGLS3d::computeDiffNorm()
{
	if (!pData->initialized)
		return 0.0f; // FIXME: Error?

	return pData->cgls.computeDiffNorm();
}



bool astraCudaFP(const float* pfVolume, float* pfProjections,
                 const CVolumeGeometry3D* pVolGeom,
                 const CProjectionGeometry3D* pProjGeom,
                 int iGPUIndex, int iDetectorSuperSampling,
                 Cuda3DProjectionKernel projKernel)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	params.iRaysPerDetDim = iDetectorSuperSampling;

	bool ok = astra::convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

	if (iDetectorSuperSampling == 0)
		return false;

	Geometry3DParameters projs = astra::convertAstraGeometry(pVolGeom, pProjGeom,
	                                                  params.volScale);


	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}


	hipPitchedPtr D_volumeData = allocateVolumeData(dims);
	ok = D_volumeData.ptr;
	if (!ok)
		return false;

	hipPitchedPtr D_projData = allocateProjectionData(dims);
	ok = D_projData.ptr;
	if (!ok) {
		hipFree(D_volumeData.ptr);
		return false;
	}

	ok &= copyVolumeToDevice(pfVolume, D_volumeData, dims, dims.iVolX);

	ok &= zeroProjectionData(D_projData, dims);

	if (!ok) {
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}

	if (projs.isParallel()) {
		switch (projKernel) {
		case ker3d_default:
			ok &= Par3DFP(D_volumeData, D_projData, dims, projs.getParallel(), params);
			break;
		case ker3d_sum_square_weights:
			ok &= Par3DFP_SumSqW(D_volumeData, D_projData, dims, projs.getParallel(), params);
			break;
		default:
			ok = false;
		}
	} else if (projs.isCone()) {
		switch (projKernel) {
		case ker3d_default:
			ok &= ConeFP(D_volumeData, D_projData, dims, projs.getCone(), params);
			break;
		default:
			ok = false;
		}
	} else {
		ok = false;
	}

	ok &= copyProjectionsFromDevice(pfProjections, D_projData,
	                                dims, dims.iProjU);


	hipFree(D_volumeData.ptr);
	hipFree(D_projData.ptr);

	return ok;

}


bool astraCudaBP(float* pfVolume, const float* pfProjections,
                 const CVolumeGeometry3D* pVolGeom,
                 const CProjectionGeometry3D* pProjGeom,
                 int iGPUIndex, int iVoxelSuperSampling)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	params.iRaysPerVoxelDim = iVoxelSuperSampling;

	bool ok = astra::convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

	Geometry3DParameters projs = astra::convertAstraGeometry(pVolGeom, pProjGeom, params.volScale);

	if (!projs.isValid())
		return false;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}


	hipPitchedPtr D_volumeData = allocateVolumeData(dims);
	ok = D_volumeData.ptr;
	if (!ok)
		return false;

	hipPitchedPtr D_projData = allocateProjectionData(dims);
	ok = D_projData.ptr;
	if (!ok) {
		hipFree(D_volumeData.ptr);
		return false;
	}

	ok &= copyProjectionsToDevice(pfProjections, D_projData,
	                              dims, dims.iProjU);

	ok &= zeroVolumeData(D_volumeData, dims);

	if (!ok) {
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}

	if (projs.isParallel())
		ok &= Par3DBP(D_volumeData, D_projData, dims, projs.getParallel(), params);
	else if (projs.isCone())
		ok &= ConeBP(D_volumeData, D_projData, dims, projs.getCone(), params);
	else
		ok = false;

	ok &= copyVolumeFromDevice(pfVolume, D_volumeData, dims, dims.iVolX);

	hipFree(D_volumeData.ptr);
	hipFree(D_projData.ptr);

	return ok;

}


// This computes the column weights, divides by them, and adds the
// result to the current volume. This is both more expensive and more
// GPU memory intensive than the regular BP, but allows saving system RAM.
bool astraCudaBP_SIRTWeighted(float* pfVolume,
                      const float* pfProjections,
                      const CVolumeGeometry3D* pVolGeom,
                      const CProjectionGeometry3D* pProjGeom,
                      int iGPUIndex, int iVoxelSuperSampling)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	params.iRaysPerVoxelDim = iVoxelSuperSampling;

	bool ok = astra::convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;


	Geometry3DParameters projs = astra::convertAstraGeometry(pVolGeom, pProjGeom, params.volScale);

	if (!projs.isValid())
		return false;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}


	hipPitchedPtr D_pixelWeight = allocateVolumeData(dims);
	ok = D_pixelWeight.ptr;
	if (!ok)
		return false;

	hipPitchedPtr D_volumeData = allocateVolumeData(dims);
	ok = D_volumeData.ptr;
	if (!ok) {
		hipFree(D_pixelWeight.ptr);
		return false;
	}

	hipPitchedPtr D_projData = allocateProjectionData(dims);
	ok = D_projData.ptr;
	if (!ok) {
		hipFree(D_pixelWeight.ptr);
		hipFree(D_volumeData.ptr);
		return false;
	}

	// Compute weights
	ok &= zeroVolumeData(D_pixelWeight, dims);
	ok &= processSino3D<opSet>(D_projData, 1.0f, dims);

	if (projs.isParallel())
		ok &= Par3DBP(D_pixelWeight, D_projData, dims, projs.getParallel(), params);
	else if (projs.isCone())
		ok &= ConeBP(D_pixelWeight, D_projData, dims, projs.getCone(), params);
	else
		ok = false;

	ok &= processVol3D<opInvert>(D_pixelWeight, dims);
	if (!ok) {
		hipFree(D_pixelWeight.ptr);
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}

	ok &= copyProjectionsToDevice(pfProjections, D_projData,
	                              dims, dims.iProjU);
	ok &= zeroVolumeData(D_volumeData, dims);
	// Do BP into D_volumeData
	if (projs.isParallel())
		ok &= Par3DBP(D_volumeData, D_projData, dims, projs.getParallel(), params);
	else if (projs.isCone())
		ok &= ConeBP(D_volumeData, D_projData, dims, projs.getCone(), params);
	else
		ok = false;

	// Multiply with weights
	ok &= processVol3D<opMul>(D_volumeData, D_pixelWeight, dims);

	// Upload previous iterate to D_pixelWeight...
	ok &= copyVolumeToDevice(pfVolume, D_pixelWeight, dims, dims.iVolX);
	if (!ok) {
		hipFree(D_pixelWeight.ptr);
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}
	// ...and add it to the weighted BP
	ok &= processVol3D<opAdd>(D_volumeData, D_pixelWeight, dims);

	// Then copy the result back
	ok &= copyVolumeFromDevice(pfVolume, D_volumeData, dims, dims.iVolX);


	hipFree(D_pixelWeight.ptr);
	hipFree(D_volumeData.ptr);
	hipFree(D_projData.ptr);

	return ok;

}

_AstraExport bool uploadMultipleProjections(CFloat32ProjectionData3D *proj,
                                         const float *data,
                                         unsigned int y_min, unsigned int y_max)
{
	assert(proj->getStorage()->isGPU());
	CDataGPU *storage = dynamic_cast<CDataGPU*>(proj->getStorage());
	astraCUDA3d::MemHandle3D hnd = storage->getHandle();

	astraCUDA3d::SDimensions3D dims1;
	dims1.iProjU = proj->getDetectorColCount();
	dims1.iProjV = proj->getDetectorRowCount();
	dims1.iProjAngles = y_max - y_min + 1;

	hipPitchedPtr D_proj = allocateProjectionData(dims1);
	bool ok = copyProjectionsToDevice(data, D_proj, dims1);
	if (!ok) {
		ASTRA_ERROR("Failed to upload projection to GPU");
		return false;
	}

	astraCUDA3d::MemHandle3D hnd1 = astraCUDA3d::wrapHandle(
			(float *)D_proj.ptr,
			dims1.iProjU, dims1.iProjAngles, dims1.iProjV,
			D_proj.pitch / sizeof(float));

	astraCUDA3d::SSubDimensions3D subdims;
	subdims.nx = dims1.iProjU;
	subdims.ny = proj->getAngleCount();
	subdims.nz = dims1.iProjV;
	subdims.pitch = D_proj.pitch / sizeof(float); // FIXME: Pitch for wrong obj!
	subdims.subnx = dims1.iProjU;
	subdims.subny = dims1.iProjAngles;
	subdims.subnz = dims1.iProjV;
	subdims.subx = 0;
	subdims.suby = y_min;
	subdims.subz = 0;

	ok = astraCUDA3d::copyIntoArray(hnd, hnd1, subdims);
	if (!ok) {
		ASTRA_ERROR("Failed to copy projection into 3d data");
		return false;
	}

	hipFree(D_proj.ptr);
	return true;
}


}
